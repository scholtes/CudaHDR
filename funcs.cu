#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

// Number of elements to put in the test array
#define TEST_SIZE 32

////////////////////////////////////////////////////////////////
////////////////// COPY EVERYTHING BELOW HERE //////////////////
////////////////////////////////////////////////////////////////

// Number of threads per block (1-d blocks)
#define BLOCK_WIDTH 8

// Perform a partial reduction 
// Only reduces per block, so this kernel may need to be called
// multiple times to generate a complete reduction
__global__ void reduceKernel(float* array,
                       const size_t array_size,
                       float (*op)(float, float))
{
  __shared__ float temp[BLOCK_WIDTH];
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int index = BLOCK_WIDTH * bx + tx;

  if(index < array_size) {
    temp[tx] = array[index];
  }

  __syncthreads();

  // Reduce
  for(int offset = BLOCK_WIDTH >> 1; offset > 0; offset >>= 1) {
    if(tx < offset) {
      temp[tx] = (*op)(temp[tx], temp[tx + offset]);
    }
    __syncthreads();
  }

  if(index < array_size) {
    array[BLOCK_WIDTH * bx] = temp[0];
  } 

}

// Create a histogram with atomics
__global__ void histogramKernel() {

}

// This performs and *exclusive* scan
__global__ void scanKernel() {

}

__device__ float addOp(float a, float b) {
  return a + b;
}

__device__ float minOp(float a, float b) {
  return a < b ? a : b;
}

__device__ float maxOp(float a, float b) {
  return a > b ? a : b;
}

// Non-kernel (host) functions

void reduce(float* d_array,
            const size_t array_size,
            float* result,
            float (*op)(float, float))
{
  float *d_array_copy;
  size_t mem_size = sizeof(float) * array_size;
  hipMalloc((void**) &d_array_copy, mem_size);
  hipMemcpy(d_array_copy, d_array, mem_size, hipMemcpyDeviceToDevice);

  size_t numBlocks = 1 + ((array_size - 1) / BLOCK_WIDTH);
  reduceKernel<<<numBlocks, BLOCK_WIDTH>>>(d_array_copy, array_size, op);

  hipMemcpy(result, d_array_copy, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_array_copy);
}

////////////////////////////////////////////////////////////////
//////////////// EXCLUDE EVERYTHING BELOW HERE /////////////////
////////////////////////////////////////////////////////////////

void generateAndCopyTestValues(float** h_A, float** d_A, size_t size) {
  unsigned int mem_size = sizeof(float) * size;
  *h_A = (float*)malloc(mem_size);
  hipMalloc((void**) d_A, mem_size);

  for(int i = 0; i < size; i++) {
    (*h_A)[i] = i+1;
  }

  hipMemcpy(*d_A, *h_A, mem_size, hipMemcpyHostToDevice);
}

void prettyprint(float *h_A, size_t size) {
  // Lots of magic numbers
  if(size <= 16) {
    for(int i = 0; i < size; i++) {
      printf("%0.1f ", h_A[i]);
    }
  } else {
    for(int i = 0; i < 8; i++) {
      printf("%0.1f ", h_A[i]);
    }
    printf("... ");
    for(int i = 0; i < 8; i++) {
      printf("%0.1f ", h_A[size +i -8]);
    }
  }
  printf("\n");
}

int main(int argc, char** argv) {
  float *h_array;
  float reduce_result_add;
  float reduce_result_max;
  float reduce_result_min;
  float *d_array;
  
  generateAndCopyTestValues(&h_array, &d_array, TEST_SIZE);

  printf("h_array = ");
  prettyprint(h_array, TEST_SIZE);

  // Perform reduce
  reduce(d_array, TEST_SIZE, &reduce_result_add, addOp);
  reduce(d_array, TEST_SIZE, &reduce_result_min, minOp);
  reduce(d_array, TEST_SIZE, &reduce_result_max, maxOp);
  printf("reduce_result_add = %0.1f\n", reduce_result_add);
  printf("reduce_result_min = %0.1f\n", reduce_result_min);
  printf("reduce_result_max = %0.1f\n", reduce_result_max);


  // Clean up
  free(h_array);
  hipFree(d_array);
  return 0;
}