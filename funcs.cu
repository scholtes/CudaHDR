
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// Number of elements to put in the test array
#define TEST_SIZE 64
#define NUM_BINS 10

////////////////////////////////////////////////////////////////
////////////////// COPY EVERYTHING BELOW HERE //////////////////
////////////////////////////////////////////////////////////////

// Number of threads per block (1-d blocks)
#define BLOCK_WIDTH 8
// Functions to reduce with
#define ADD 0
#define MIN 1
#define MAX 2
// Device functions

__device__ float addOp(float a, float b) {
  return a + b;
}

__device__ float minOp(float a, float b) {
  return a < b ? a : b;
}

__device__ float maxOp(float a, float b) {
  return a > b ? a : b;
}

// Perform a partial reduction 
// Only reduces per block, so this kernel may need to be called
// multiple times to generate a complete reduction
__global__ void reduceKernel(float* array,
                       const size_t array_size,
                       const unsigned int op,
                       const size_t step)
{
  __shared__ float temp[BLOCK_WIDTH];
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int index = BLOCK_WIDTH * bx + tx;

  if(index < array_size) {
    temp[tx] = array[index * step];
  }

  __syncthreads();

  // Reduce
  for(int offset = BLOCK_WIDTH >> 1; offset > 0; offset >>= 1) {
    if(tx < offset) {
        switch(op) {
        case ADD:
          temp[tx] = addOp(temp[tx], temp[tx + offset]);
          break;
        case MIN:
          temp[tx] = minOp(temp[tx], temp[tx + offset]);
          break;
        case MAX:
          temp[tx] = maxOp(temp[tx], temp[tx + offset]);
          break;
        default:
          break;
        }
    }
    __syncthreads();
  }

  if(index < array_size) {
    array[BLOCK_WIDTH * bx] = temp[0];
  } 

}

// Create a histogram with atomics
__global__ void histogramKernel(unsigned int* d_hist,
                                const float* const d_array,
                                const size_t array_size,
                                float max,
                                float min,
                                const size_t numBins)
{
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int index = BLOCK_WIDTH * bx + tx;
  float range = max - min;

  // Initialize temp
  if(index < numBins) {
    d_hist[index] = 0;
  }

  __syncthreads();

  if(index < array_size) {
    size_t bin = (size_t)((d_array[index] - min) * numBins / range );
    atomicAdd(&d_hist[bin], 1);
  }
}

// This performs and *exclusive* scan
__global__ void scanKernel() {

}

void reduce(float* d_array,
            const size_t array_size,
            float* result,
            unsigned int op)
{
  float *d_array_copy;
  size_t mem_size = sizeof(float) * array_size;
  hipMalloc((void**) &d_array_copy, mem_size);
  hipMemcpy(d_array_copy, d_array, mem_size, hipMemcpyDeviceToDevice);

  // First pass: 
  size_t numBlocks = 1 + ((array_size - 1) / BLOCK_WIDTH);
  reduceKernel<<<numBlocks, BLOCK_WIDTH>>>(d_array_copy, array_size, op, 1);
  // Second pass:
  reduceKernel<<<1, BLOCK_WIDTH>>>(d_array_copy, array_size, op, BLOCK_WIDTH);


  hipMemcpy(result, d_array_copy, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_array_copy);
}

//d_hist, d_array, TEST_SIZE, reduce_result_max, reduce_result_min, numBins
void histogram(unsigned int** d_hist,
               const float* const d_array,
               const size_t array_size,
               float max,
               float min,
               const size_t numBins)
{
  hipMalloc((void**) d_hist, sizeof(unsigned int) * numBins);

  size_t numBlocks = 1 + ((array_size - 1) / BLOCK_WIDTH);
  histogramKernel<<<numBlocks, BLOCK_WIDTH>>>(
      *d_hist,
      d_array,
      array_size,
      max,
      min,
      numBins
  );
}

////////////////////////////////////////////////////////////////
//////////////// EXCLUDE EVERYTHING BELOW HERE /////////////////
////////////////////////////////////////////////////////////////

void generateAndCopyTestValues(float** h_A, float** d_A, size_t size) {
  unsigned int mem_size = sizeof(float) * size;
  *h_A = (float*)malloc(mem_size);
  hipMalloc((void**) d_A, mem_size);

  for(int i = 0; i < size; i++) {
    (*h_A)[i] = i+1;
  }

  hipMemcpy(*d_A, *h_A, mem_size, hipMemcpyHostToDevice);
}

void prettyprint(float *h_A, size_t size) {
  // Lots of magic numbers
  if(size <= 16) {
    for(int i = 0; i < size; i++) {
      printf("%0.1f ", h_A[i]);
    }
  } else {
    for(int i = 0; i < 8; i++) {
      printf("%0.1f ", h_A[i]);
    }
    printf("... ");
    for(int i = 0; i < 8; i++) {
      printf("%0.1f ", h_A[size +i -8]);
    }
  }
  printf("\n");
}

void prettyprint(unsigned int *h_A, size_t size) {
  // Lots of magic numbers
  if(size <= 16) {
    for(int i = 0; i < size; i++) {
      printf("%d ", h_A[i]);
    }
  } else {
    for(int i = 0; i < 8; i++) {
      printf("%d ", h_A[i]);
    }
    printf("... ");
    for(int i = 0; i < 8; i++) {
      printf("%d ", h_A[size +i -8]);
    }
  }
  printf("\n");
}

int main(int argc, char** argv) {
  // Reduce
  float *h_array;
  float *d_array;
  float reduce_result_add;
  float reduce_result_max;
  float reduce_result_min;
  // Histogram
  unsigned int *h_hist;
  unsigned int *d_hist;
  size_t numBins = NUM_BINS;
  
  generateAndCopyTestValues(&h_array, &d_array, TEST_SIZE);

  printf("h_array = ");
  prettyprint(h_array, TEST_SIZE);

  // Perform reduce
  reduce(d_array, TEST_SIZE, &reduce_result_add, ADD);
  reduce(d_array, TEST_SIZE, &reduce_result_min, MIN);
  reduce(d_array, TEST_SIZE, &reduce_result_max, MAX);
  printf("reduce_result_add = %0.1f\n", reduce_result_add);
  printf("reduce_result_min = %0.1f\n", reduce_result_min);
  printf("reduce_result_max = %0.1f\n", reduce_result_max);

  // Perform histogram
  histogram(&d_hist, d_array, TEST_SIZE, reduce_result_max, reduce_result_min, numBins);
  // Host histogram (not to be used in student_func)
  h_hist = (unsigned int*)malloc(sizeof(unsigned int) * numBins);
  hipMemcpy(h_hist, d_hist, sizeof(unsigned int) * numBins, hipMemcpyDeviceToHost);
  printf("h_hist = ");
  prettyprint(h_hist, numBins);


  // Clean up
  free(h_array);
  free(h_hist);
  hipFree(d_array);
  hipFree(d_hist);
  return 0;
}