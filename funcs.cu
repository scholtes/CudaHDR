
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// Number of elements to put in the test array
#define TEST_SIZE 16
#define NUM_BINS 10

////////////////////////////////////////////////////////////////
////////////////// COPY EVERYTHING BELOW HERE //////////////////
////////////////////////////////////////////////////////////////

// Number of threads per block (1-d blocks)
#define BLOCK_WIDTH 4
// Functions to reduce with
#define ADD 0
#define MIN 1
#define MAX 2
// Device functions

__device__ float addOp(float a, float b) {
  return a + b;
}

__device__ float minOp(float a, float b) {
  return a < b ? a : b;
}

__device__ float maxOp(float a, float b) {
  return a > b ? a : b;
}

// Perform a partial reduction 
// Only reduces per block, so this kernel may need to be called
// multiple times to generate a complete reduction
__global__ void reduceKernel(float* array,
                       const size_t array_size,
                       const unsigned int op,
                       const size_t step)
{
  __shared__ float temp[BLOCK_WIDTH];
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int index = BLOCK_WIDTH * bx + tx;

  if(index < array_size) {
    temp[tx] = array[index * step];
  }

  __syncthreads();

  // Reduce
  for(int offset = BLOCK_WIDTH >> 1; offset > 0; offset >>= 1) {
    if(tx < offset) {
        switch(op) {
        case ADD:
          temp[tx] = addOp(temp[tx], temp[tx + offset]);
          break;
        case MIN:
          temp[tx] = minOp(temp[tx], temp[tx + offset]);
          break;
        case MAX:
          temp[tx] = maxOp(temp[tx], temp[tx + offset]);
          break;
        default:
          break;
        }
    }
    __syncthreads();
  }

  if(index < array_size) {
    array[BLOCK_WIDTH * bx] = temp[0];
  } 

}

// Create a histogram with atomics
__global__ void histogramKernel(unsigned int* d_hist,
                                const float* const d_array,
                                const size_t array_size,
                                float max,
                                float min,
                                const size_t numBins)
{
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int index = BLOCK_WIDTH * bx + tx;
  float range = max - min;

  // Initialize temp
  if(index < numBins) {
    d_hist[index] = 0;
  }

  __syncthreads();

  if(index < array_size) {
    size_t bin = (size_t)((d_array[index] - min) * numBins / range );
    atomicAdd(&d_hist[bin], 1);
  }
}

// This performs a partial exclusive scan (blockwise) using Blelloch's method
__global__ void scanKernel(unsigned int* d_cdf,
                           unsigned int* d_input,
                           const size_t array_size)
{
  __shared__ unsigned int temp[BLOCK_WIDTH<<1];
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int index = BLOCK_WIDTH * bx + tx;
  int offset = 1;

  if(2*index + 1 < array_size) {
    temp[2*index] = d_input[2*index];
    temp[2*index + 1] = d_input[2*index + 1];
  }
  
  // Up-sweep
  for(int powOf2 = (2*BLOCK_WIDTH)>>1; powOf2 > 0; powOf2 >>= 1) {
    __syncthreads();
    if(tx < powOf2) {
      int idx1 = offset*(2*tx + 1) - 1 + 2*BLOCK_WIDTH*bx;
      int idx2 = offset*(2*tx + 2) - 1 + 2*BLOCK_WIDTH*bx;
      temp[idx2] += temp[idx1];
    }
    offset <<= 1;
  }

  __syncthreads();
  ///// The below will need to be remembered for multiple blocks /////
  if(tx == 0) {
    temp[2*BLOCK_WIDTH*(bx + 1) - 1] = 0;
  }

  // Down-sweep
  for(int powOf2 = 1; powOf2 < 2*BLOCK_WIDTH; powOf2 <<= 1) {
    offset >>= 1;
    __syncthreads();
    if(tx < powOf2) {
      int idx1 = offset*(2*tx + 1) - 1 + 2*BLOCK_WIDTH*bx;
      int idx2 = offset*(2*tx + 2) - 1 + 2*BLOCK_WIDTH*bx;
      unsigned int t = temp[idx1];
      temp[idx1] = temp[idx2];
      temp[idx2] += t;
    }
  }

  __syncthreads();
  if(2*index + 1 < array_size) {
    d_cdf[2*index] = temp[2*index];
    d_cdf[2*index + 1] = temp[2*index + 1];
  }
}

void reduce(float* d_array,
            const size_t array_size,
            float* result,
            unsigned int op)
{
  float *d_array_copy;
  size_t mem_size = sizeof(float) * array_size;
  hipMalloc((void**) &d_array_copy, mem_size);
  hipMemcpy(d_array_copy, d_array, mem_size, hipMemcpyDeviceToDevice);

  // First pass: 
  size_t numBlocks = 1 + ((array_size - 1) / BLOCK_WIDTH);
  reduceKernel<<<numBlocks, BLOCK_WIDTH>>>(d_array_copy, array_size, op, 1);
  // Second pass:
  reduceKernel<<<1, BLOCK_WIDTH>>>(d_array_copy, array_size, op, BLOCK_WIDTH);


  hipMemcpy(result, d_array_copy, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_array_copy);
}

//d_hist, d_array, TEST_SIZE, reduce_result_max, reduce_result_min, numBins
void histogram(unsigned int** d_hist,
               const float* const d_array,
               const size_t array_size,
               float max,
               float min,
               const size_t numBins)
{
  hipMalloc((void**) d_hist, sizeof(unsigned int) * numBins);

  size_t numBlocks = 1 + ((array_size - 1) / BLOCK_WIDTH);
  histogramKernel<<<numBlocks, BLOCK_WIDTH>>>(
      *d_hist,
      d_array,
      array_size,
      max,
      min,
      numBins
  );
}

void scan(unsigned int** d_cdf,
          unsigned int* d_input,
          const size_t array_size)
{
  hipMalloc((void**) d_cdf, sizeof(unsigned int) * array_size);

  // Note the divide by 2 (a block can handle array of size 2*BLOCK_WIDTH)
  size_t numBlocks = (1 + ((array_size - 1) / BLOCK_WIDTH))/2;
  scanKernel<<<numBlocks, BLOCK_WIDTH>>>(*d_cdf, d_input, array_size);
}

////////////////////////////////////////////////////////////////
//////////////// EXCLUDE EVERYTHING BELOW HERE /////////////////
////////////////////////////////////////////////////////////////

void generateAndCopyTestValues(float** h_A, float** d_A, size_t size) {
  unsigned int mem_size = sizeof(float) * size;
  *h_A = (float*)malloc(mem_size);
  hipMalloc((void**) d_A, mem_size);

  for(int i = 0; i < size; i++) {
    (*h_A)[i] = i+1;
  }

  hipMemcpy(*d_A, *h_A, mem_size, hipMemcpyHostToDevice);
}

void genTestValsCDF(unsigned int** h_A, unsigned int** d_A, size_t size) {
  unsigned int mem_size = sizeof(unsigned int) * size;
  *h_A = (unsigned int*)malloc(mem_size);
  hipMalloc((void**) d_A, mem_size);

  for(int i = 0; i < size; i++) {
    (*h_A)[i] = i+1;
  }

  hipMemcpy(*d_A, *h_A, mem_size, hipMemcpyHostToDevice);
}

void prettyprint(float *h_A, size_t size) {
  // Lots of magic numbers
  if(size <= 16) {
    for(int i = 0; i < size; i++) {
      printf("%0.1f ", h_A[i]);
    }
  } else {
    for(int i = 0; i < 8; i++) {
      printf("%0.1f ", h_A[i]);
    }
    printf("... ");
    for(int i = 0; i < 8; i++) {
      printf("%0.1f ", h_A[size +i -8]);
    }
  }
  printf("\n");
}

void prettyprint(unsigned int *h_A, size_t size) {
  // Lots of magic numbers
  if(size <= 16) {
    for(int i = 0; i < size; i++) {
      printf("%d ", h_A[i]);
    }
  } else {
    for(int i = 0; i < 8; i++) {
      printf("%d ", h_A[i]);
    }
    printf("... ");
    for(int i = 0; i < 8; i++) {
      printf("%d ", h_A[size +i -8]);
    }
  }
  printf("\n");
}

int main(int argc, char** argv) {
  // Reduce
  float *h_array;
  float *d_array;
  float reduce_result_add;
  float reduce_result_max;
  float reduce_result_min;
  // Histogram
  unsigned int *h_hist;
  unsigned int *d_hist;
  size_t numBins = NUM_BINS;
  // CDF (exclusive scan - prefix sum)
  unsigned int *h_toBeScanned;
  unsigned int *d_toBeScanned;
  unsigned int *h_cdf;
  unsigned int *d_cdf;
  
  generateAndCopyTestValues(&h_array, &d_array, TEST_SIZE);

  printf("h_array = ");
  prettyprint(h_array, TEST_SIZE);

  // Perform reduce
  reduce(d_array, TEST_SIZE, &reduce_result_add, ADD);
  reduce(d_array, TEST_SIZE, &reduce_result_min, MIN);
  reduce(d_array, TEST_SIZE, &reduce_result_max, MAX);
  printf("reduce_result_add = %0.1f\n", reduce_result_add);
  printf("reduce_result_min = %0.1f\n", reduce_result_min);
  printf("reduce_result_max = %0.1f\n", reduce_result_max);

  // Perform histogram
  histogram(&d_hist, d_array, TEST_SIZE, reduce_result_max, reduce_result_min, numBins);
  // Host histogram (not to be used in student_func)
  h_hist = (unsigned int*)malloc(sizeof(unsigned int) * numBins);
  hipMemcpy(h_hist, d_hist, sizeof(unsigned int) * numBins, hipMemcpyDeviceToHost);
  printf("h_hist = ");
  prettyprint(h_hist, numBins);

  // Set up and perform exclusive scan
  genTestValsCDF(&h_toBeScanned, &d_toBeScanned, TEST_SIZE);

  printf("\nh_toBeScanned = ");
  prettyprint(h_toBeScanned, TEST_SIZE);

  scan(&d_cdf, d_toBeScanned, TEST_SIZE);
  h_cdf = (unsigned int*)malloc(sizeof(unsigned int) * TEST_SIZE);
  hipMemcpy(h_cdf, d_cdf, sizeof(unsigned int) * TEST_SIZE, hipMemcpyDeviceToHost);
  printf("h_cdf = ");
  prettyprint(h_cdf, TEST_SIZE);


  // Clean up
  free(h_array);
  free(h_hist);
  free(h_cdf);
  free(h_toBeScanned);
  hipFree(d_array);
  hipFree(d_hist);
  hipFree(d_cdf);
  hipFree(d_toBeScanned);
  return 0;
}